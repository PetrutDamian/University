﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <fstream>
#include <time.h> 
using namespace std;


__global__ void blur(int* flat, int* result, int lines, int cols, int channels, int scale) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < lines * cols * channels) {
        int line = tid / (cols * channels);
        int rest = tid % (cols * channels);
        int col = rest / channels;
        int channel = rest % channels;

        int sum = 0;
        int nr = 0;
        for (int i = 0 - scale / 2;i <= scale / 2;i++)
            for (int j = 0 - scale / 2;j <= scale / 2;j++) {
                int line2 = line + i;
                int col2 = col + j;
                if (line2 >= 0 && line2 < lines && col2 >= 0 && col2 < cols) { // check if pixel is outside of frame
                    nr++;
                    int index = line2 * cols * channels + col2 * channels + channel;
                    sum += flat[index];
                }

            }
        result[tid] = sum / nr;
    }
}

void blurSecvential(int* flat, int* result, int lines, int cols, int channels, int scale) {
    for (int i = 0;i < lines * cols * channels;i++) {
        int line = i / (cols * channels);
        int rest = i % (cols * channels);
        int col = rest / channels;
        int channel = rest % channels;

        int sum = 0;
        int nr = 0;
        for (int i = 0 - scale / 2;i <= scale / 2;i++) {
            for (int j = 0 - scale / 2;j <= scale / 2;j++) {
                int line2 = line + i;
                int col2 = col + j;
                if (line2 >= 0 && line2 < lines && col2 >= 0 && col2 < cols) { // check if pixel is outside of frame
                    nr++;
                    int index = line2 * cols * channels + col2 * channels + channel;
                    sum += flat[index];
                }
            }
        }
        result[i] = sum / nr;
    }
}


int* flatten(int*** matrix, int lines, int cols, int channels) {
    int* flat = (int*)malloc(lines * cols * channels * sizeof(int));
    int id = 0;
    for (int i = 0; i < lines; i++) {
        for (int j = 0; j < cols; j++) {
            for (int c = 0; c < channels; c++) {
                flat[id] = matrix[i][j][c];
                id++;
            }
        }
    }
    return flat;
}
int*** unflatten(int* arr, int lines, int cols, int channels) {
    int*** img = (int***)malloc(lines * sizeof(int**));
    int id = 0;

    for (int i = 0; i < lines; i++) {
        img[i] = (int**)malloc(cols * sizeof(int*));

        for (int j = 0; j < cols; j++) {
            img[i][j] = (int*)malloc(channels * sizeof(int));

            for (int c = 0; c < channels; c++) {
                img[i][j][c] = arr[id];
                id++;
            }
        }
    }
    return img;
}

int main()
{
    string name = "cake2";
    string readCmd = "python read.py " + name + ".png";
    string writeCmd = "python write.py " + name + "blur.png";

    int scale = 45;
    const char* command = readCmd.c_str(); 
    system(command);

    //read matrix

    ifstream fin("pixels.txt");
    int lines, cols, channels;
    fin >> lines >> cols >> channels;
    int*** matrix = (int***)malloc(lines * sizeof(int**));
    for (int i = 0; i < lines; i++) {

        matrix[i] = (int**)malloc(cols * sizeof(int*));
        for (int j = 0; j < cols; j++) {
            int* rgb = (int*)malloc(channels * sizeof(int));
            fin >> rgb[0] >> rgb[1] >> rgb[2];
            matrix[i][j] = rgb;
        }
    }
    fin.close();
    int* flat = flatten(matrix, lines, cols, channels);
    int size = lines * cols * channels;
    int* result = (int*)malloc(size * sizeof(int));

    int* deviceMatrix, * deviceResult;

    clock_t begin = clock();
    hipMalloc(&deviceMatrix, size * sizeof(int));
    hipMalloc(&deviceResult, size * sizeof(int));
    hipMemcpy(
        deviceMatrix, flat,
        size * sizeof(int),
        hipMemcpyHostToDevice
    );
    int num_threads = 1 <<10;
    int num_blocks = (size + num_threads - 1) / num_threads;

    blur <<<num_blocks, num_threads >>> (deviceMatrix, deviceResult, lines, cols, channels, scale);

    hipMemcpy(
        result, deviceResult,
        size * sizeof(int),
        hipMemcpyDeviceToHost
    );
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    int*** resultMatrix = unflatten(result, lines, cols, channels);

    ofstream out("pixels2.txt");
    out << lines << " " << cols << " " << channels << "\n";
    for (int i = 0; i < lines; i++) {
        for (int j = 0; j < cols; j++) {
            for (int k = 0; k < channels; k++) {
                out << resultMatrix[i][j][k] << " ";
            }
            out << "\n";
        }
    }
    out.close();

    command = writeCmd.c_str();
    system(command);
    printf("Paralel:  %f seconds", time_spent);

    begin = clock();
    blurSecvential(flat, result, lines, cols, channels, scale);
    end = clock();
    resultMatrix = unflatten(result, lines, cols, channels);

    ofstream fout("pixels2.txt");
    fout << lines << " " << cols << " " << channels << "\n";
    for (int i = 0; i < lines; i++) {
        for (int j = 0; j < cols; j++) {
            for (int k = 0; k < channels; k++) {
                fout << resultMatrix[i][j][k] << " ";
            }
            fout << "\n";
        }
    }
    fout.close();


    writeCmd = "python write.py " + name + "blur_secv.png";
    command = writeCmd.c_str();
    system(command);
    time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Secvential:  %f seconds", time_spent);
    return 0;
}